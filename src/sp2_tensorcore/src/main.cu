#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <math.h>
#include <fstream>
#include <regex>
#include <typeinfo>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <random>
#include <ctime>
#include <cmath>
#include <vector>
#include <chrono>
#include "../include/tcore_hp_emulator.cuh"
#include "../include/linalg_tools.cuh"
#include "../include/prg_sp2_tensorcore.cuh"



void produce_hamiltonian (const unsigned N, float *X) {
    for(int i=0; i<N; ++i) {
        for(int j=i; j<N; ++j) {
            X[i+j*N] = exp(-0.5f*abs((float)(i-j)))*sin((float)(i+1));
            X[j+i*N] = X[i+j*N];
        }
    }
};


int main(int argc, char *argv[])
{

    // Matrix size
    size_t N = atoi(argv[1]);
    size_t Nocc = atoi(argv[2]);

    int Stopp = 0;
    int Kvot = 0;
    int iter = 0;
    int Pur_Start = 0;
    float eps = 1e-16;


    std::vector<float> Idemp_Error;
    
    std::cout << "Mat Size: " << N << std::endl;
    std::cout << "Occupied orbitals: " << Nocc << std::endl;

    // Set GPU
    int device = 0;
    hipSetDevice(device);

    // Cublas Handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    

    hipblasStatus_t cublasStat = hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    float *H;
    H = (float*) malloc(N*N*sizeof(float));
    double *D; 
    D = (double*) malloc(N*N*sizeof(double));
    

    // Produce Hamiltonian and Identity matrix 
    std::cout << "Loading Hamiltonian..." << std::endl;
    produce_hamiltonian(N,H);
    
    // Get device id
    hipGetDevice(&device); 

    float idemtol=1e-16;
    char sp2conv;
    int verbose=0;
    float bndfil=float(Nocc)/float(N);
    std::cout << N << Nocc << bndfil << std::endl;
    prg_sp2_tensorcore(5000,H,D,eps,.1,1,1000,sp2conv,idemtol,verbose);
    
}



